#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <random>
#include <cmath>
#include <fstream>
#include <sstream>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <>
#include <stdio.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <string>
#include "GeometryVector.h"
#include "kiss_fft.h"
#include "kiss_fftnd.h"

void computeGridSize(int n, int blockSize, int &numBlocks, int &numThreads);
int iDivUp(int a, int b);


__global__ void energy_kernal(int nGridperside, double * E_xy, double * E_xx, double * E_buffer_xy, double * E_buffer_xx, int * pindex, double * energytotal, int num2check)
{

	int index = blockIdx.x*blockDim.x + threadIdx.x; // this is the parallel index, each thread correspond to a single grid location

	if (index >= nGridperside*nGridperside) return; // just to be safe

    int ix = index / nGridperside;
    int iy = index % nGridperside;
    int bufferCenter = nGridperside/2;

    double exx = E_xx[index];
    double exy = E_xy[index];

    double energy_old = exx * exx + exy * exy;

    for (int ip=0; ip < num2check ; ip++)   // loop through all the sites to check
    {
        int gid = pindex[ip];

    	int rx = gid / nGridperside;
        int ry = gid % nGridperside;

        int xInBuffer = bufferCenter - rx + ix;
    	while (xInBuffer < 0)
          xInBuffer += nGridperside;
    	while (xInBuffer >= nGridperside)
          xInBuffer -= nGridperside;

	    int yInBuffer = bufferCenter - ry + iy;
		while (yInBuffer < 0)
		  yInBuffer += nGridperside;
		while (yInBuffer >= nGridperside)
		  yInBuffer -= nGridperside;


    double dexx = E_buffer_xx[xInBuffer * nGridperside + yInBuffer];
    double dexy = E_buffer_xy[xInBuffer * nGridperside + yInBuffer];

    if (ry != iy || rx != ix)
    {
         double e_inc = (exx + dexx) * (exx + dexx) + (exy + dexy) * (exy + dexy) - energy_old;
//         energytotal[index] = e_inc;
         atomicAdd(&energytotal[ip], e_inc);   // a slow reduction method to ensure no core race
    }else
    {
    	 double e_dec = -energy_old;
//    	 energytotal[index] = e_dec;
         atomicAdd(&energytotal[ip], e_dec);
    }

    }
}

template <typename T>
void writebinary(const std::vector<T> &data, int nGridPerSide)
{
  std::ofstream re_data;
  re_data.open ("data_hasRe.bin", std::ios::out | std::ios::binary | std::fstream::app);

  int nsum = nGridPerSide*nGridPerSide;
  int total_re = 0;
  for (int i = 0; i < nsum; i++)
  {
      if (data[i]==1){
          total_re +=1;
      }
  }
  re_data.write((char*)&total_re,sizeof(int));

  for (int i = 0; i < nsum; i++)
  {

        if (data[i]==1){
          int gindex=i;
          re_data.write((char*)&gindex,sizeof(int));
         }

  }
  std::cout<< "rearrangement written: " << total_re <<std::endl;
  re_data.close();

}

template <typename T>
void writebinary_scalar(const std::vector<T> &data, int nGridPerSide)
{
  std::ofstream re_data;
  re_data.open ("data_Softness.bin", std::ios::out | std::ios::binary | std::fstream::app);

  double nsum = nGridPerSide*nGridPerSide;
  re_data.write((char*)&nsum,sizeof(double));

  for (int i = 0; i < nsum; i++)
  {

          double gindex=data[i];
          re_data.write((char*)&gindex,sizeof(double));

  }
  std::cout<< "softness written: " << nsum <<std::endl;
  re_data.close();
}

void writebinary_test(double * data, int nGridPerSide)
{
  std::ofstream re_data;
  re_data.open ("data_test.bin", std::ios::out | std::ios::binary | std::fstream::app);

  double nsum = nGridPerSide*nGridPerSide;
  re_data.write((char*)&nsum,sizeof(double));

  for (int i = 0; i < nsum; i++)
  {

          double gindex=data[i];
//	      double gindex = i;
          re_data.write((char*)&gindex,sizeof(double));

  }
  std::cout<< "test written: " << nsum <<std::endl;
  re_data.close();
}


class gridModel
{
public:
    int nGridPerSide;
    double lGrid;
    int bufferCenter;
    double mean_softness;
    double dmean_softness;

    std::vector<double> dSBuffer, alls;
    std::vector<GeometryVector> dEBuffer;
    std::vector<char> rearrangingStep;
    std::vector<bool> hasRearranged;

    std::mt19937 rEngine;
    std::normal_distribution<double> sDistribution;
    std::normal_distribution<double> eDistribution;

//    std::gamma_distribution<double> coeffDistribution;
    std::normal_distribution<double> coeffDistribution;
    std::vector<double> yieldStrainCoeff;

//    float *hPos;
//    float *dPos;

    double *dEbuffer_xy;
    double *dEbuffer_xx;
    double *dE_xy;
    double *dE_xx;
    double *denergy;

    double *hEbuffer_xy;
    double *hEbuffer_xx;
    double *hE_xy;
    double *hE_xx;
    double *henergy;




    gridModel(int nGrid, double lGrid) : rEngine(0), eDistribution(0.0, 0.01), sDistribution(-2.0, 2.0), coeffDistribution(1.5, 0.667), nGridPerSide(nGrid), lGrid(lGrid)
    {
    }

    bool startRearranging(double e1, double e2, double s, int i)
    {
        double yieldStrain = 0.07 - 0.01 * s;
        if (yieldStrain < 0.05)
            yieldStrain = 0.05;
        yieldStrain*=yieldStrainCoeff[i];

        double mod2 = e1*e1 + e2*e2;

        return mod2 > yieldStrain * yieldStrain;
    }

    double dsFromRearranger(double dx, double dy, double r)
    {
        if (r < 4.0)
            return -0.03;
        else if (r < 30)
            return 1.0 / r / r / r - 0.16 / r / r * std::sin(2.0 * std::atan2(dy, dx));
        else
            return 0.0;
    }

    void getBuffer()  //assume effect from all rearrangements are the same, so only need once
    {
        bufferCenter = nGridPerSide / 2;
        dEBuffer.resize(nGridPerSide * nGridPerSide); // vector of doubles
        dSBuffer.resize(nGridPerSide * nGridPerSide);
        for (int i = 0; i < nGridPerSide; i++)
            for (int j = 0; j < nGridPerSide; j++)
            {
                double dx = (i - bufferCenter) * lGrid;
                double dy = (j - bufferCenter) * lGrid;
                double r = std::sqrt(dx * dx + dy * dy);
                int index = i * nGridPerSide + j;
                // dEBuffer[index] = eFromRearranger(dx, dy, r);
                dSBuffer[index] = dsFromRearranger(dx, dy, r);
            }

        double factor;
        {
            //strain buffer calculated by Fourier transform
            int numPixels = nGridPerSide * nGridPerSide;
            kiss_fft_cpx *inbuf = new kiss_fft_cpx[numPixels];
            kiss_fft_cpx *outbuf = new kiss_fft_cpx[numPixels];
            //fill in inbuf
            for (int i = 0; i < nGridPerSide; i++)
                for (int j = 0; j < nGridPerSide; j++)
                {
                    int ii = (i > bufferCenter) ? i - nGridPerSide : i;
                    int jj = (j > bufferCenter) ? j - nGridPerSide : j;
                    double L = nGridPerSide * lGrid;
                    double pm = 2 * M_PI * ii / L;
                    double qn = 2 * M_PI * jj / L;
                    double q2 = pm * pm + qn * qn;

                    inbuf[i * nGridPerSide + j].r = -4 * pm * pm * qn * qn / q2 / q2;
                    inbuf[i * nGridPerSide + j].i = 0;
                }
            inbuf[0].r = 0;

            int temp[2] = {nGridPerSide, nGridPerSide};
            kiss_fftnd_cfg st = kiss_fftnd_alloc(temp, 2, true, nullptr, nullptr);
            kiss_fftnd(st, inbuf, outbuf);
            //fill in dEBuffer
            factor = 0.02 / std::fabs(outbuf[0].r);

            for (int i = 0; i < nGridPerSide; i++)
                for (int j = 0; j < nGridPerSide; j++)
                {
                    int index = i * nGridPerSide + j;
                    int ii = i - bufferCenter;
                    while (ii < 0)
                        ii += nGridPerSide;
                    int jj = j - bufferCenter;
                    while (jj < 0)
                        jj += nGridPerSide;
                    int index2 = ii * nGridPerSide + jj;
                    dEBuffer[index2] = GeometryVector(outbuf[index].r * factor, 0.0);
                }

            free(st);
            delete[] outbuf;
            delete[] inbuf;
        }
        {
            //strain buffer calculated by Fourier transform, another component
            int numPixels = nGridPerSide * nGridPerSide;
            kiss_fft_cpx *inbuf = new kiss_fft_cpx[numPixels];
            kiss_fft_cpx *outbuf = new kiss_fft_cpx[numPixels];
            //fill in inbuf
            for (int i = 0; i < nGridPerSide; i++)
                for (int j = 0; j < nGridPerSide; j++)
                {
                    int ii = (i > bufferCenter) ? i - nGridPerSide : i;
                    int jj = (j > bufferCenter) ? j - nGridPerSide : j;
                    double L = nGridPerSide * lGrid;
                    double pm = 2 * M_PI * ii / L;
                    double qn = 2 * M_PI * jj / L;
                    double q2 = pm * pm + qn * qn;

                    inbuf[i * nGridPerSide + j].r = -2 * pm * qn * (pm * pm - qn * qn) / q2 / q2;
                    inbuf[i * nGridPerSide + j].i = 0;
                }
            inbuf[0].r = 0;

            int temp[2] = {nGridPerSide, nGridPerSide};
            kiss_fftnd_cfg st = kiss_fftnd_alloc(temp, 2, true, nullptr, nullptr);
            kiss_fftnd(st, inbuf, outbuf);
            //fill in dEBuffer

            for (int i = 0; i < nGridPerSide; i++)
                for (int j = 0; j < nGridPerSide; j++)
                {
                    int index = i * nGridPerSide + j;
                    int ii = i - bufferCenter;
                    while (ii < 0)
                        ii += nGridPerSide;
                    int jj = j - bufferCenter;
                    while (jj < 0)
                        jj += nGridPerSide;
                    int index2 = ii * nGridPerSide + jj;
                    dEBuffer[index2].x[1] = outbuf[index].r * factor;
                }

            free(st);
            delete[] outbuf;
            delete[] inbuf;
        }
    }

    void initialize()
    {
        int nSite = nGridPerSide * nGridPerSide;
        alls.resize(nSite);
        hasRearranged.resize(nSite);
        rearrangingStep.resize(nSite);
        yieldStrainCoeff.resize(nSite);

        // initialize host arrays
        hEbuffer_xy = new double[nSite];
        hEbuffer_xx = new double[nSite];
        hE_xy = new double[nSite];
        hE_xx = new double[nSite];
        henergy = new double[1];



        memset(hEbuffer_xy, 0, nSite*sizeof(double));
        memset(hEbuffer_xx, 0, nSite*sizeof(double));
        memset(hE_xy, 0, nSite*sizeof(double));
        memset(hE_xx, 0, nSite*sizeof(double));
        memset(henergy, 0, sizeof(double));

        this->getBuffer();

        for (int i = 0; i < nSite; i++)
        {

            this->hE_xy[i] = this->eDistribution(this->rEngine);
            this->hE_xx[i] = this->eDistribution(this->rEngine);

            this->alls[i] = this->sDistribution(this->rEngine); //initialize softness with a distrbution s
            this->yieldStrainCoeff[i] = this->coeffDistribution(this->rEngine);
            this->hasRearranged[i] = 0;
            this->rearrangingStep[i] = 0;

            // this is a duplicate, probably don't need both
            this->hEbuffer_xy[i] = this->dEBuffer[i].x[0];
            this->hEbuffer_xx[i] = this->dEBuffer[i].x[1];

        }

//        writebinary_test( hEbuffer_xy , nGridPerSide);
//        writebinary_test( hEbuffer_xx , nGridPerSide);


        // initialize and fill the device buffer
        size_t bytes_E = nSite*sizeof(double);

        hipMalloc(&dEbuffer_xy, bytes_E);
        hipMalloc(&dEbuffer_xx, bytes_E);
        hipMalloc(&dE_xy, bytes_E);
        hipMalloc(&dE_xx, bytes_E);
        hipMalloc(&denergy, sizeof(double));

        hipMemcpy(dEbuffer_xy, hEbuffer_xy, bytes_E, hipMemcpyHostToDevice);
        hipMemcpy(dEbuffer_xx, hEbuffer_xx, bytes_E, hipMemcpyHostToDevice);
        hipMemcpy(dE_xy, hE_xy, bytes_E, hipMemcpyHostToDevice);
        hipMemcpy(dE_xx, hE_xx, bytes_E, hipMemcpyHostToDevice);
        hipMemcpy(denergy, henergy, sizeof(double), hipMemcpyHostToDevice);
    }


    void shear()
    {

        int nSite = nGridPerSide * nGridPerSide;
#pragma omp parallel for schedule(static)
        for (int i = 0; i < nSite; i++)
        {

            this->hE_xy[i] += 1.0e-6;
            this->hE_xx[i] += 0.0e-6;

            this->alls[i] += dmean_softness;
            this->hasRearranged[i] = 0;
            this->rearrangingStep[i] = 0;
        }
    }


    int avalanche(std::string outputPrefix = "")
    {
        int nSite = nGridPerSide * nGridPerSide;
//        double deltaEnergy;
        int numRearrange = 1;

#pragma omp parallel
        {
            while (numRearrange > 0)
            {
#pragma omp for schedule(static)
                for (int i = 0; i < nSite; i++)
                    if (startRearranging(hE_xy[i],hE_xx[i], alls[i],i))
                    {
                        rearrangingStep[i] = 1;
                    }

#pragma omp barrier
#pragma omp single
          {
//        	  // initiate a thrust vector to collect energy change for all potential site
              thrust::host_vector<double> h_energy(nSite);
              thrust::fill(h_energy.begin(),h_energy.end(), 0.0);
              thrust::device_vector<double> d_energy = h_energy;
              double *d_energy_pt = thrust::raw_pointer_cast( &d_energy[0] );

              // initiate a thrust vector to collect the index of all potential site
              thrust::host_vector<int> h_pindex(nSite);
              thrust::fill(h_pindex.begin(),h_pindex.end(), 0);
              thrust::device_vector<int> d_pindex = h_pindex;
              int *d_pindex_pt = thrust::raw_pointer_cast( &d_pindex[0] );

              // collect the index of all potential sites
              int site_2_check=0;
              for (int i = 0; i < nSite; i++)
              {
            	  if (rearrangingStep[i] > 0){
            		  d_pindex[site_2_check] = i;
            		  site_2_check+=1;
            	  }

              }
              std::cout<< "total potential site checked: " << site_2_check << std::endl;

              size_t bytes_E = nSite*sizeof(double);
              hipMemcpy(dE_xy, hE_xy, bytes_E, hipMemcpyHostToDevice);
              hipMemcpy(dE_xx, hE_xx, bytes_E, hipMemcpyHostToDevice);

              //stop rearrangements that increases energy

              // the idea is to assign each grid position to a gpu-thread, within each grid position, we loop and calculate energy change for all potential sites
              int numBlocks, numThreads;
              int nSite = nGridPerSide * nGridPerSide;
              computeGridSize(nSite, 256, numBlocks, numThreads);

              energy_kernal <<< numBlocks, numThreads >>> (nGridPerSide, dE_xy, dE_xx, dEbuffer_xy, dEbuffer_xx, d_pindex_pt, d_energy_pt, site_2_check);

              hipDeviceSynchronize();

              for (int i = 0; i < site_2_check; i++)
              {
//            	  std::cout << "energy change: " << d_energy[i] << std::endl;
            	  if (d_energy[i]>0){
            		  int index_grid = d_pindex[i];
            		  rearrangingStep[index_grid] = 0;
            	  }

              }
            }


#pragma omp barrier
                //rearrangement affect other sites parameters
//                numRearrange = 0;
                for (int i = 0; i < nSite; i++)
                {
                    if (rearrangingStep[i] > 0)
                    {
                        //update softness and strain
                        int rx = i / nGridPerSide;
                        int ry = i % nGridPerSide;
#pragma omp for schedule(static)
                    for (int x = 0; x < nGridPerSide; x++)
                    {
                        int xInBuffer = bufferCenter - rx + x;
                        while (xInBuffer < 0)
                            xInBuffer += nGridPerSide;
                        while (xInBuffer >= nGridPerSide)
                            xInBuffer -= nGridPerSide;
                        for (int y = 0; y < nGridPerSide; y++)
                        {
                            int yInBuffer = bufferCenter - ry + y;
                             while (yInBuffer < 0)
                                 yInBuffer += nGridPerSide;
                             while (yInBuffer >= nGridPerSide)
                                 yInBuffer -= nGridPerSide;
                            hE_xx[x * nGridPerSide + y] +=  hEbuffer_xx[xInBuffer * nGridPerSide + yInBuffer];
                            hE_xy[x * nGridPerSide + y] +=  hEbuffer_xy[xInBuffer * nGridPerSide + yInBuffer];

                            alls[x * nGridPerSide + y] += dSBuffer[xInBuffer * nGridPerSide + yInBuffer];
                        }
                    }
                    }
                }

#pragma omp single
                {
                	numRearrange=0;

                    for (int i = 0; i < nSite; i++)
                    {
                        if (rearrangingStep[i] > 0)
                        {
                            //carry out the rearrangement
                            rearrangingStep[i]++;
                            hasRearranged[i] = 1;
                            hE_xy[i] = 0.0;
                            hE_xx[i] = 0.0;

                            alls[i] = sDistribution(rEngine);
                            yieldStrainCoeff[i] = coeffDistribution(rEngine);

                            numRearrange++;
                        }
                    }
                    std::cout << "Internal loop, number of rearrangement = " << numRearrange << std::endl;
                }
            }
        }

        int numRe_frame = 0;
        for (int i = 0; i < nSite; i++)
        {
            if (hasRearranged[i] == 1)
            {
                numRe_frame+=1;
            }

        }
        return numRe_frame;
    }
};

int main()
{
    const int nGridPerSide = 300;
    gridModel model(nGridPerSide, 1.0);
    model.initialize();
    int numAvalanche = 0;
    int strainstep = 0;
    double meanS = 0.0;
    std::cout<<"initialized"<<std::endl;
    int total_re=0;
//
    while (strainstep<10000000 && meanS<1.0)
    {
        model.shear();

        std::stringstream ss;
        ss << "avalanche_" << numAvalanche;

        int numRe = model.avalanche(ss.str());

        if (numRe>0)
        {

        double sum = 0.0;
        for (auto &s : model.alls)
           sum += s;

        meanS = sum / model.alls.size();

        writebinary<bool>(model.hasRearranged, nGridPerSide);

        //this part write the softness field
//        strainstep+=1;
//        if ((strainstep-1)%50==0)
//        {
//            writebinary_scalar<double>(model.alls, nGridPerSide);
//        }

        // this part output the mean softness and mean energy
//        std::ofstream re_mean_data;
//        re_mean_data.open ("data_mean.bin", std::ios::out | std::ios::binary | std::fstream::app);
//        double nstep = strainstep;
//        re_mean_data.write((char*)&nstep,sizeof(double));
//        re_mean_data.write((char*)&meanS,sizeof(double));
//
//        double etotal=0;
//        for (int i = 0; i < nGridPerSide*nGridPerSide; i++)
//         {
//          etotal += model.hE_xx[i] * model.hE_xx[i] + model.hE_xy[i] * model.hE_xy[i];
//         }
//
//        re_mean_data.write((char*)&etotal,sizeof(double));
//        re_mean_data.close();

        total_re = total_re+1;

//        std::cout << "Currently at step: " << strainstep << ", Number of rearrangement:"<<  numRe;
//        std::cout << ", mean s=" << sum / model.alls.size() << ", total energy=" << etotal << std::endl;
        }
        strainstep+=1;
        std::cout << "Step: " << strainstep << ", Num rearrangement: "<<  numRe << "  total avalanches: " << total_re << std::endl;

    }
}



//leave them alone, these are for determine how to assign threads
void computeGridSize(int n, int blockSize, int &numBlocks, int &numThreads)
{
    numThreads = std::min(blockSize, n);
    numBlocks = iDivUp(n, numThreads);
}

int iDivUp(int a, int b)
{
    return (a % b != 0) ? (a / b + 1) : (a / b);
}
